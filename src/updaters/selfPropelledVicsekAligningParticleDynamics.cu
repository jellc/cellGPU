#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "selfPropelledVicsekAligningParticleDynamics.cuh"

/** \file selfPropelledVicsekAligningParticleDynamics.cu
    * Defines kernel callers and kernels for GPU calculations of simple active 2D cell models
*/

/*!
    \addtogroup simpleEquationOfMotionKernels
    @{
*/

/*!
Each thread calculates the displacement of an individual cell
*/
__global__ void spp_vicsek_aligning_eom_integration_kernel(Dscalar2 *forces,
                                           Dscalar2 *velocities,
                                           Dscalar2 *displacements,
                                           Dscalar2 *motility,
                                           Dscalar *cellDirectors,
                                           int *nNeighbors,
                                           int *neighbors,
                                           Index2D  n_idx,
                                           hiprandState *RNGs,
                                           int N,
                                           Dscalar deltaT,
                                           int Timestep,
                                           Dscalar mu,
                                           Dscalar Eta)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    //get an appropriate random angle displacement
    hiprandState_t randState;
    randState=RNGs[idx];
    Dscalar v0 = motility[idx].x;
    //Dscalar Dr = motility[idx].y;
    Dscalar randomAngle = 2.0*PI*hiprand_uniform(&randState);
    RNGs[idx] = randState;

    Dscalar currentTheta = cellDirectors[idx];
    //update displacements
    velocities[idx].x = v0*Cos(currentTheta) + mu*forces[idx].x;
    velocities[idx].y = v0*Sin(currentTheta) + mu*forces[idx].y;
    displacements[idx] = deltaT*velocities[idx];

    Dscalar2 direction; direction.x = 0.0; direction.y=0.0;
    int neigh = nNeighbors[idx];
    for (int nn =0; nn < neigh; ++nn)
        {
        Dscalar curTheta = cellDirectors[neighbors[n_idx(nn,idx)]];
        direction.x += Cos(curTheta);
        direction.y += Sin(curTheta);
        }
    direction.x += neigh*Eta*Cos(randomAngle);
    direction.y += neigh*Eta*Sin(randomAngle);
    Dscalar phi = atan2(direction.y,direction.x);
    
    //update director
    cellDirectors[idx] = phi;

    return;
    };

//!get the current timesteps vector of displacements into the displacement vector
bool gpu_spp_vicsek_aligning_eom_integration(
                    Dscalar2 *forces,
                    Dscalar2 *velocities,
                    Dscalar2 *displacements,
                    Dscalar2 *motility,
                    Dscalar *cellDirectors,
                    int *nNeighbors,
                    int *neighbors,
                    Index2D  &n_idx,
                    hiprandState *RNGs,
                    int N,
                    Dscalar deltaT,
                    int Timestep,
                    Dscalar mu,
                    Dscalar Eta)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    spp_vicsek_aligning_eom_integration_kernel<<<nblocks,block_size>>>(
                                forces,velocities,displacements,motility,cellDirectors,
                                nNeighbors,neighbors,n_idx,
                                RNGs,
                                N,deltaT,Timestep,mu, Eta);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
